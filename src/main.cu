#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <cassert>
#include <vector>

#define TILE_DIM 32
#define BLOCK_DIM 32

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)


//mat1 m x n
//mat2 n x p
//out m x p
template<typename T>
__global__ void maxPlusMulKernel(const T *mat1, const T *mat2, T *out,
                                 const size_t m, const size_t n, const size_t p) {
    const auto i{blockIdx.y * blockDim.y + threadIdx.y};
    const auto j{blockIdx.x * blockDim.x + threadIdx.x};
    if ((i >= m) || (j >= p))
        return;
    out[i * p + j] = mat1[i * n] + mat2[j];
    for (size_t k{1}; k < n; k++)
        out[i * p + j] = max(out[i * p + j], mat1[i * n + k] + mat2[k * p + j]);

}


//mat1 m x n
//mat2 n x p
//out m x p
template<typename T>
__global__ void optimizedMaxPlusMulKernel(const T *mat1, const T *mat2, T *out,
                                          const size_t m, const size_t n, const size_t p) {
    __shared__ T mat1Tile[TILE_DIM][TILE_DIM];
    __shared__ T mat2Tile[TILE_DIM][TILE_DIM];
    T val{0};
    
   for (size_t tile_idx{0};
         tile_idx < ceilf(static_cast<float>(n) / BLOCK_DIM); ++tile_idx)
    {
        size_t i{blockIdx.y * blockDim.y + threadIdx.y};
        size_t j{tile_idx * blockDim.x + threadIdx.x};
        if ((i < m) && (j < n))
        {
            mat1Tile[threadIdx.y][threadIdx.x] = mat1[i * n + j];
        }
        else
        {
            mat1Tile[threadIdx.y][threadIdx.x] = 0;
        }
        i = tile_idx * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if ((i < n) && (j < p))
        {
            mat2Tile[threadIdx.y][threadIdx.x] = mat2[i * p + j];
        }
        else
        {
            mat2Tile[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (size_t k{0}; k < BLOCK_DIM; ++k)
        {
            val = max(val, mat1Tile[threadIdx.y][k] + mat2Tile[k][threadIdx.x]);
        }
        __syncthreads();
    }

    size_t i{blockIdx.y * blockDim.y + threadIdx.y};
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};

    if ((i < m) && (j < p))
    {
        out[i * p + j] = val;
    }

}

//mat1 m x n
//mat2 m x n
//out m x n
template<typename T>
__global__ void maxPlusAddKernel(const T *mat1, const T *mat2, T *out,
                                 const size_t m, const size_t n) {
    const auto i{blockIdx.y * blockDim.y + threadIdx.y};
    const auto j{blockIdx.x * blockDim.x + threadIdx.x};
    if ((i > m) || (j > n))
        return;
    out[i + j] = max(mat1[i + j], mat2[i + j]);
}

//mat1 m x n
//mat2 m x n
//out m x n
template<typename T>
__global__ void optimizedMaxPlusAddKernel(const T *mat1, const T *mat2, T *out,
                                          const size_t m, const size_t n) {

}


//mat1 m x n
//mat2 n x p
//out m x p
template<typename T>
__global__ void minPlusMulKernel(const T *mat1, const T *mat2, T *out,
                                 const size_t m, const size_t n, const size_t p) {
    const auto i{blockIdx.y * blockDim.y + threadIdx.y};
    const auto j{blockIdx.x * blockDim.x + threadIdx.x};
    if ((i >= m) || (j >= p))
        return;
    out[i * p + j] = mat1[i * n] + mat2[j];
    for (size_t k{1}; k < n; k++)
        out[i * p + j] = min(out[i * p + j], mat1[i * n + k] + mat2[k * p + j]);
}


//mat1 m x n
//mat2 n x p
//out m x p
template<typename T>
__global__ void optimizedMinPlusMulKernel(const T *mat1, const T *mat2, T *out,
                                          const size_t m, const size_t n, const size_t p) {

}


//mat1 m x n
//mat2 m x n
//out m x n
template<typename T>
__global__ void minPlusAddKernel(const T *mat1, const T *mat2, T *out,
                                 const size_t m, const size_t n) {
    const auto i{blockIdx.y * blockDim.y + threadIdx.y};
    const auto j{blockIdx.x * blockDim.x + threadIdx.x};
    if ((i > m) || (j > n))
        return;
    out[i + j] = min(mat1[i + j], mat2[i + j]);
}


//mat1 m x n
//mat2 m x n
//out m x n
template<typename T>
__global__ void optimizedMinPlusAddKernel(const T *mat1, const T *mat2, T *out,
                                          const size_t m, const size_t n) {

}


template<typename T>
void showMtr(const T *vec, size_t size) {
    for (int i = 0; i < size; ++i)
        std::cout << vec[i] << " ";
    std::cout << std::endl;
}

template <typename T>
void mm_cuda(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n,
             size_t p,
             void (*fun)(T const*, T const*, T*, size_t, size_t, size_t))
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    fun<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3, m, n, p);
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Tropic Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}


int main() {

//    torch::Tensor tensor = torch::rand([2, 3]);
//    std::cout << tensor << std::endl;
    const int m = 3, n = 2, p = 3;

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate host memory

    a = (float *) malloc(sizeof(float) * m * n);
    b = (float *) malloc(sizeof(float) * n * p);
    out = (float *) malloc(sizeof(float) * m * p);

    // Initialize host arrays
    for (int i = 0; i < m * n; i++) {
        a[i] = float(i) + 1;
    }
    for (int i = 0; i < n * p; i++) {
        b[i] = float(i) + 1;
    }
//    for (int i = 0; i < m * p; i++) {
//        out[i] = float(INT_MIN);
//    }

    // Allocate device memory

    CUDA_CHECK(hipMalloc((void **) &d_a, sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc((void **) &d_b, sizeof(float) * n * p));
    CUDA_CHECK(hipMalloc((void **) &d_out, sizeof(float) * m * p));

    // Transfer data from host to device memory
    CUDA_CHECK(hipMemcpy(d_a, a, sizeof(float) * m * n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, sizeof(float) * n * p, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_out, out, sizeof(float) * m * p, hipMemcpyHostToDevice));

    // Executing kernel
    dim3 blocks_per_grid(5);
    dim3 threads_per_block(TILE_DIM, TILE_DIM);
//    minPlusMulKernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out, m, n, p);
    //maxPlusMulKernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out, m, n, p);
    //plusMulKernel<float><<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out, m, n, p);

    //maxPlusMulKernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out,
    //                                                                m, n, p);
    
    mm_cuda(d_a, d_b, d_out, m, n, p, optimizedMaxPlusMulKernel);
    //minPlusMulKernel<><<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out, m, n, p);

    //maxPlusAddKernel<float><<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out,
//                                                                    m, n);

    CUDA_CHECK(hipMemcpy(out, d_out, sizeof(float) * m * p, hipMemcpyDeviceToHost));

    showMtr(out, m * n);
    //printf("PASSED\n");
    std::cout << "PASSED" << std::endl;
    // Deallocate device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_out));

    // Deallocate host memory
    free(a);
    free(b);
    free(out);
}
