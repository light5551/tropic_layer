#include <cstdio>
#include <stdlib.h>
#include <limits.h>
//#include <cuda.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 32

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)


// cpu
//    for (size_t i = 0; i < m; i++) {
//        for (size_t j = 0; j < p; j++) {
//            for (size_t k = 0; k < n; k++) {
//                float tmp = mat1[i * n + k] * mat2[k * p + j];
//                out[i * p + j] = max(tmp, out[i * p + j]);
//            }
//        }
//    }

//mat1 m x n
//mat2 n x p
//out m x p
__global__ void maxPlusMulKernel(const float *mat1, const float *mat2, float *out,
                                 const int m, const int n, const int p) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i >= m) || (j >= p))
        return;
    for (size_t k = 0; k < n; k++)
        out[i * p + j] = max(out[i * p + j], mat1[i * n + k] + mat2[k * p + j]);
}

//mat1 m x n
//mat2 n x p
//out m x p
__global__ void minPlusMulKernel(const float *mat1, const float *mat2, float *out,
                                 const int m, const int n, const int p) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i >= m) || (j >= p))
        return;
    for (size_t k = 0; k < n; k++)
        out[i * p + j] = min(out[i * p + j], mat1[i * n + k] + mat2[k * p + j]);
}


void showMtr(const float * vec, int size){
    for (int i = 0; i < size; ++i)
        printf("%3.f ", vec[i]);
    printf("\n");
}

int main() {

    const int m = 3, n = 2, p = 2;

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate host memory

    a = (float *) malloc(sizeof(float) * m*n);
    b = (float *) malloc(sizeof(float) * n*p);
    out = (float *) malloc(sizeof(float) * m*p);

    // Initialize host arrays
    for (int i = 0; i < m*n; i++) {
        a[i] = float(i) + 1;
    }
    for (int i = 0; i < n*p; i++) {
        b[i] = float(i) + 1;
    }
    for (int i = 0; i < m*p; i++) {
        out[i] = float(INT_MIN);
    }

    // Allocate device memory
    CUDA_CHECK(hipMalloc((void **) &d_a, sizeof(float) * m*n));
    CUDA_CHECK(hipMalloc((void **) &d_b, sizeof(float) * n*p));
    CUDA_CHECK(hipMalloc((void **) &d_out, sizeof(float) * m*p));

    // Transfer data from host to device memory
    CUDA_CHECK(hipMemcpy(d_a, a, sizeof(float) * m*n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, sizeof(float) * n*p, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_out, out, sizeof(float) * m*p, hipMemcpyHostToDevice));

    // Executing kernel
    dim3 blocks_per_grid(1);
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
//    minPlusMulKernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out, m, n, p);
    maxPlusMulKernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_out, m, n, p);
    CUDA_CHECK(hipMemcpy(out, d_out, sizeof(float) * m*p, hipMemcpyDeviceToHost));

    showMtr(out, m*p);
    printf("PASSED\n");

    // Deallocate device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_out));

    // Deallocate host memory
    free(a);
    free(b);
    free(out);
}